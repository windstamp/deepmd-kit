#include "hip/hip_runtime.h"
#if GOOGLE_CUDA
#include "gpu_cuda.h"
#elif PADDLE_HIP
#include "gpu_hip.h"
#endif

#include "prod_virial.h"

template<typename FPTYPE>
__global__ void virial_deriv_wrt_neighbors_a(
    FPTYPE * virial, 
    FPTYPE * atom_virial,
    const FPTYPE * net_deriv,
    const FPTYPE * in_deriv,
    const FPTYPE * rij,
    const int * nlist,
    const int nloc,
    const int nnei) 
{
  // idx -> nloc
  // idy -> nnei
  // idz = dd0 * 3 + dd1
  // dd0 = idz / 3
  // dd1 = idz % 3
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int idy = blockIdx.y;
  const unsigned int idz = threadIdx.y;
  const unsigned int idw = threadIdx.z;
  const int ndescrpt = nnei * 4;
  if (idx >= nloc) {
      return;
  }
  int j_idx = nlist[idx * nnei + idy];
  if (j_idx < 0) {
      return;
  }
  // atomicAdd(
  //    virial + idz, 
  //    net_deriv[idx * ndescrpt + idy * 4 + idw] * rij[idx * nnei * 3 + idy * 3 + idz / 3] * in_deriv[idx * ndescrpt * 3 + (idy * 4 + idw) * 3 + idz % 3]);
  atomicAdd(
      atom_virial + j_idx * 9 + idz, 
      net_deriv[idx * ndescrpt + idy * 4 + idw] * rij[idx * nnei * 3 + idy * 3 + idz % 3] * in_deriv[idx * ndescrpt * 3 + (idy * 4 + idw) * 3 + idz / 3]);
}

template<typename FPTYPE>
__global__ void virial_deriv_wrt_neighbors_r(
    FPTYPE * virial, 
    FPTYPE * atom_virial,
    const FPTYPE * net_deriv,
    const FPTYPE * in_deriv,
    const FPTYPE * rij,
    const int * nlist,
    const int nloc,
    const int nnei) 
{
    // idx -> nloc
    // idy -> nnei
    // idz = dd0 * 3 + dd1
    // dd0 = idz / 3
    // dd1 = idz % 3
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int idy = blockIdx.y;
    const unsigned int idz = threadIdx.y;
    const int ndescrpt = nnei * 1;

    if (idx >= nloc) {
        return;
    }
    int j_idx = nlist[idx * nnei + idy];
    if (j_idx < 0) {
        return;
    }
    // atomicAdd(
    //    virial + idz, 
    //    net_deriv[idx * ndescrpt + idy * 4 + idw] * rij[idx * nnei * 3 + idy * 3 + idz / 3] * in_deriv[idx * ndescrpt * 3 + (idy * 4 + idw) * 3 + idz % 3]);
    atomicAdd(
        atom_virial + j_idx * 9 + idz, 
        net_deriv[idx * ndescrpt + idy] * rij[idx * nnei * 3 + idy * 3 + idz % 3] * in_deriv[idx * ndescrpt * 3 + idy * 3 + idz / 3]);
}

namespace deepmd {
template<typename FPTYPE>
void prod_virial_a_gpu_cuda(
    FPTYPE * virial, 
    FPTYPE * atom_virial, 
    const FPTYPE * net_deriv, 
    const FPTYPE * in_deriv, 
    const FPTYPE * rij, 
    const int * nlist, 
    const int nloc, 
    const int nall, 
    const int nnei)
{
  #if GOOGLE_CUDA
  cudaErrcheck(hipMemset(
      virial, 
      0.0, sizeof(FPTYPE) * 9));
  cudaErrcheck(hipMemset(
      atom_virial, 
      0.0, sizeof(FPTYPE) * 9 * nall));
  #elif PADDLE_HIP
  hipErrcheck(hipMemset(
      virial, 
      0.0, sizeof(FPTYPE) * 9));
  hipErrcheck(hipMemset(
      atom_virial, 
      0.0, sizeof(FPTYPE) * 9 * nall));
  #endif

  const int LEN = 16;
  int nblock = (nloc + LEN -1) / LEN;
  dim3 block_grid(nblock, nnei);
  dim3 thread_grid(LEN, 9, 4);

  // compute virial of a frame
  #if GOOGLE_CUDA
  virial_deriv_wrt_neighbors_a<<<block_grid, thread_grid>>>(
      virial, atom_virial, 
      net_deriv, in_deriv, rij, nlist, nloc, nnei);
  #elif PADDLE_HIP
  hipLaunchKernelGGL(virial_deriv_wrt_neighbors_a, block_grid, thread_grid, 0, 0,
      virial, atom_virial, 
      net_deriv, in_deriv, rij, nlist, nloc, nnei);
  #endif
}

template<typename FPTYPE>
void prod_virial_r_gpu_cuda(
    FPTYPE * virial, 
    FPTYPE * atom_virial, 
    const FPTYPE * net_deriv, 
    const FPTYPE * in_deriv, 
    const FPTYPE * rij, 
    const int * nlist, 
    const int nloc, 
    const int nall, 
    const int nnei)
{
  #if GOOGLE_CUDA
  cudaErrcheck(hipMemset(
      virial, 
      0.0, sizeof(FPTYPE) * 9));
  cudaErrcheck(hipMemset(
      atom_virial, 
      0.0, sizeof(FPTYPE) * 9 * nall));
  #elif PADDLE_HIP
  hipErrcheck(hipMemset(
      virial, 
      0.0, sizeof(FPTYPE) * 9));
  hipErrcheck(hipMemset(
      atom_virial, 
      0.0, sizeof(FPTYPE) * 9 * nall));
  #endif

  const int LEN = 16;
  int nblock = (nloc + LEN -1) / LEN;
  dim3 block_grid(nblock, nnei);
  dim3 thread_grid(LEN, 9);

  // compute virial of a frame
  #if GOOGLE_CUDA
  virial_deriv_wrt_neighbors_r<<<block_grid, thread_grid>>>(
      virial, atom_virial, 
      net_deriv, in_deriv, rij, nlist, nloc, nnei);
  #elif PADDLE_HIP
  hipLaunchKernelGGL(virial_deriv_wrt_neighbors_r, block_grid, thread_grid, 0, 0,
      virial, atom_virial, 
      net_deriv, in_deriv, rij, nlist, nloc, nnei);
  #endif
}

template void prod_virial_a_gpu_cuda<float>(float * virial, float * atom_virial, const float * net_deriv, const float * in_deriv, const float * rij, const int * nlist, const int nloc, const int nall, const int nnei);
template void prod_virial_a_gpu_cuda<double>(double * virial, double * atom_virial, const double * net_deriv, const double * in_deriv, const double * rij, const int * nlist, const int nloc, const int nall, const int nnei);
template void prod_virial_r_gpu_cuda<float>(float * virial, float * atom_virial, const float * net_deriv, const float * in_deriv, const float * rij, const int * nlist, const int nloc, const int nall, const int nnei);
template void prod_virial_r_gpu_cuda<double>(double * virial, double * atom_virial, const double * net_deriv, const double * in_deriv, const double * rij, const int * nlist, const int nloc, const int nall, const int nnei);
}
